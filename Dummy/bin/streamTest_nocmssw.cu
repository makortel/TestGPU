#include "hip/hip_runtime.h"
#include <stdio.h>
#include <future>
#include <thread>
#include <chrono>
#include <iostream>

__constant__ int factor = 0;

__global__ 
void vectorAdd(int *a, int *b, int *c) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    c[i] = factor*(a[i] + b[i]);
}

__global__
void matrixAdd(int **a,int **b, int**c) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    c[i][j] = a[i][j] + b[i][j];
}

#define PRINT(x) \
    std::cout << #x " = " << x << std::endl

void func(const char* ptr) {
    std::cout << "ptr = " << ptr << std::endl;
}

#define N 1024*1024
#define FULL_DATA_SIZE N*20

__global__ void kernel(int *a, int *b, int *c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int idx1 = (idx+1) % 256;
        int idx2 = (idx+2) % 256;
        float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}

int main(int argc, char** argv) {
    // start time
    auto startTime = std::chrono::high_resolution_clock::now();
    printf("Hello World\n");

    // get the number of devices
    int numDevices;
    hipGetDeviceCount(&numDevices);
    PRINT(numDevices);

    hipDeviceProp_t prop;
    for (auto i=0 ; i<numDevices; i++) {
        hipGetDeviceProperties(&prop, i);
        PRINT(prop.name);
        PRINT(prop.totalGlobalMem);
        PRINT(prop.sharedMemPerBlock);
        PRINT(prop.regsPerBlock);
        PRINT(prop.warpSize);
        PRINT(prop.memPitch);
        PRINT(prop.maxThreadsPerBlock);
        PRINT(prop.maxThreadsDim[0]);
        PRINT(prop.maxThreadsDim[1]);
        PRINT(prop.maxThreadsDim[2]);
        PRINT(prop.maxGridSize[0]);
        PRINT(prop.maxGridSize[1]);
        PRINT(prop.maxGridSize[2]);
        PRINT(prop.totalConstMem);
        PRINT(prop.major);
        PRINT(prop.minor);
        PRINT(prop.clockRate);
        PRINT(prop.textureAlignment);
        PRINT(prop.deviceOverlap);
        PRINT(prop.multiProcessorCount);
        PRINT(prop.kernelExecTimeoutEnabled);
        PRINT(prop.integrated);
        PRINT(prop.canMapHostMemory);
        PRINT(prop.computeMode);
        PRINT(prop.maxTexture1D);
        PRINT(prop.maxTexture2D[0]);
        PRINT(prop.maxTexture2D[1]);
        PRINT(prop.maxTexture3D[0]);
        PRINT(prop.maxTexture3D[1]);
        PRINT(prop.maxTexture3D[2]);
//        PRINT(prop.maxTexture2DArray[0]);
//        PRINT(prop.maxTexture2DArray[1]);
//        PRINT(prop.maxTexture2DArray[2]);
        PRINT(prop.concurrentKernels);
    }

    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipStream_t stream;
    hipStreamCreate(&stream);

    int *h_a, *h_b, *h_c;
    int *d_a, *d_b, *d_c;

    hipMalloc((void**)&d_a, N * sizeof(int));
    hipMalloc((void**)&d_b, N * sizeof(int));
    hipMalloc((void**)&d_c, N * sizeof(int));

    hipHostAlloc((void**)&h_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&h_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&h_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

    for (auto i =0; i<FULL_DATA_SIZE; i++) {
        h_a[i] = i;
        h_b[i] = i*i;
    }

    for (auto i=0; i<FULL_DATA_SIZE; i+=N) {
        hipMemcpyAsync(d_a, h_a + i, N * sizeof(int),
                        hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_b, h_b + i, N * sizeof(int),
                        hipMemcpyHostToDevice, stream);

        kernel<<<N/256, 256, 0, stream>>>(d_a, d_b, d_c);

        hipMemcpyAsync(h_c + i, d_c, N * sizeof(int),
                        hipMemcpyDeviceToHost, stream);
    }

    // CPU to wait until GPU has finished
    hipStreamSynchronize(stream);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time taken: %3.1f ms\n", elapsedTime);
    for (auto i=0; i<10; i++)
        printf("c[%d] = %d\n", i, h_c[i]);

    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    hipStreamDestroy(stream);

    // stop time
    auto stopTime = std::chrono::high_resolution_clock::now();
    PRINT((stopTime - startTime).count());

    printf("Goodbye World\n");
}
