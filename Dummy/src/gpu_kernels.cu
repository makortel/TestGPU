#include "hip/hip_runtime.h"
#include "TestGPU/Dummy/interface/gpu_kernels.h"

#include <stdio.h>

#define NUM_VALUES 10000

//
// Vector Addition Kernel
//
template<typename T>
__global__
void vectorAdd(T *a, T *b, T *c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

namespace testgpu {

//
// The following macros to simplify the template instantiation
//
#define ALLOCATE(NUM_OF_VALUES, TYPE) \
    template void allocate<NUM_OF_VALUES, TYPE>(TYPE**)
#define COPY(NUM_OF_VALUES, TYPE) \
    template void copy<NUM_OF_VALUES, TYPE>(TYPE*, TYPE*, bool)
#define WRAPPERVECTORADD(NUM_OF_VALUES, TYPE) \
    template void wrapperVectorAdd<NUM_OF_VALUES, TYPE>(TYPE*, TYPE*, TYPE*)
#define RELEASE(TYPE) \
    template void release<TYPE>(TYPE*)

template<int NUM_OF_VALUES, typename T>
void allocate(T** values) {
    hipMalloc(values, NUM_OF_VALUES*sizeof(T));
}

// FIXME: can be put into a separate file
ALLOCATE(10000, int);

template<int NUM_OF_VALUES, typename T>
void copy(T* h_values, T* d_values, bool direction) {
    if (direction) 
        hipMemcpy(d_values, h_values, NUM_OF_VALUES*sizeof(T), hipMemcpyHostToDevice);
    else
        hipMemcpy(h_values, d_values, NUM_OF_VALUES*sizeof(T), hipMemcpyDeviceToHost);
}

// FIXME: can be put into a separate file
COPY(10000, int);

template<int NUM_OF_VALUES, typename T>
void wrapperVectorAdd(T* d_a, T* d_b, T* d_c) {
    int threadsPerBlock {256};
    int blocksPerGrid = (NUM_OF_VALUES + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c);
}

// FIXME: can be put into a separate file
WRAPPERVECTORADD(10000, int);

template<typename T>
void release(T* d_values) {
    hipFree(d_values);
}

// FIXME: can be put into a separate file
RELEASE(int);

//
// Standalone function that allocates/copies/launches/frees and prints the results
//
void launch_on_gpu() {
    printf("start launch_on_gpu\n");
    int h_a[NUM_VALUES], h_b[NUM_VALUES], h_c[NUM_VALUES];
    for (auto i=0; i<NUM_VALUES; i++) {
        h_a[i] = i;
        h_b[i] = i*i;
    }

    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, NUM_VALUES*sizeof(int));
    hipMalloc(&d_b, NUM_VALUES*sizeof(int));
    hipMalloc(&d_c, NUM_VALUES*sizeof(int));

    hipMemcpy(d_a, h_a, NUM_VALUES*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, NUM_VALUES*sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock {256};
    int blocksPerGrid = (NUM_VALUES + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c);

    hipMemcpy(h_c, d_c, NUM_VALUES*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    for (auto i=0; i<10; i++) {
        printf("c[%d] = %d\n", i, h_c[i]);
    }

    printf("\n");
    printf("stop launch_on_gpu\n");
}

}
