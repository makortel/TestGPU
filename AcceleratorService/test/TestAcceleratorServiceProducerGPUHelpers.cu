#include "TestAcceleratorServiceProducerGPUHelpers.h"

int TestAcceleratorServiceProducerGPUHelpers_simple_kernel(int input) {
  // Example from Viktor
  constexpr int NUM_VALUES = 10000;
  
  int h_a[NUM_VALUES], h_b[NUM_VALUES], h_c[NUM_VALUES];
  for (auto i=0; i<NUM_VALUES; i++) {
    h_a[i] = i;
    h_b[i] = i*i;
  }

  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, NUM_VALUES*sizeof(int));
  hipMalloc(&d_b, NUM_VALUES*sizeof(int));
  hipMalloc(&d_c, NUM_VALUES*sizeof(int));

  hipMemcpy(d_a, h_a, NUM_VALUES*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, NUM_VALUES*sizeof(int), hipMemcpyHostToDevice);

  int threadsPerBlock {256};
  int blocksPerGrid = (NUM_VALUES + threadsPerBlock - 1) / threadsPerBlock;
  vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c);

  hipMemcpy(h_c, d_c, NUM_VALUES*sizeof(int), hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  int ret = 0;
  for (auto i=0; i<10; i++) {
    ret += h_c[i];
  }

  return ret;
}

